#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// the predicate here is divisibility by 3
__global__ void mapPredicate(int* in, int* out, unsigned int dataSize)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > dataSize) return;

	out[id] = 0;
	if (in[id] % 3 == 0) {
		out[id] = 1;
	}
}

__global__ void scan(int* in, int* out, unsigned int dataSize)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > dataSize) return;
	out[id] = in[id];

	__syncthreads();
	for (int d = 1, d_pow = 1; d < (log2f(dataSize) + 1); d++) {
		if (id + 1 > d_pow) {
			out[id] += out[id - d_pow];
		}
		d_pow *= 2;
		__syncthreads();
	}
}

__global__ void compact(int* in, int* predicates, int* offsets, int* out, unsigned int dataSize)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > dataSize) return;

	if (predicates[id] == 1) {
		out[offsets[id] - 1] = in[id];
	}
}

hipError_t withCuda(int* compacted, const int* numbers, unsigned int size)
{
	int* dev_numbers = 0;
	int* dev_predicates = 0;
	int* dev_offsets = 0;
	int* dev_compacted = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system
	cudaStatus = hipSetDevice(0);

	// Allocate GPU buffers for three vectors (two input, one output
	cudaStatus = hipMalloc((void**)&dev_compacted, size * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_numbers, size * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_predicates, size * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_offsets, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_numbers, numbers, size * sizeof(int), hipMemcpyHostToDevice);
	mapPredicate << <1, size >> > (dev_numbers, dev_predicates, size);
	scan << <1, size >> > (dev_predicates, dev_offsets, size);
	compact << <1, size >> > (dev_numbers, dev_predicates, dev_offsets, dev_compacted, size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(compacted, dev_compacted, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_numbers);
	hipFree(dev_predicates);
	hipFree(dev_offsets);
	hipFree(dev_compacted);

	return cudaStatus;
}

int main()
{
	const int arraySize = 1024;
	int* numbers = (int*)malloc(arraySize * sizeof(int));
	int* compacted = (int*)malloc(arraySize * sizeof(int));

	for (int i = 0; i < arraySize; i++) {
		numbers[i] = i + 1;
	}


	hipError_t cudaStatus = withCuda(compacted, numbers, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "withCuda failed!");
		return 1;
	}

	const int max = fmin(1000, arraySize);

	printf("numbers:   ");
	for (int i = 0; i < max - 1; i++) {
		printf("%4d, ", numbers[i]);
	}
	printf("%4d ", numbers[max - 1]);

	printf("\ncompacted: ");
	for (int i = 0; i < max - 1; i++) {
		printf("%4d, ", compacted[i]);
	}
	printf("%4d ", compacted[max - 1]);


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	free(numbers);
	free(compacted);

	return 0;
}
