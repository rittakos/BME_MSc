#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void evenOddSort(int* a, int size) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = 0; i < size; i += 2) {
        if (id + i < size - 1) {
            if (a[id + i] > a[id + i + 1]) {
                int temp = a[id + i];
                a[id + i] = a[id + i + 1];
                a[id + i + 1] = temp;
            }
        }
        __syncthreads();

        if (id + i + 1 < size - 1) {
            if (a[id + i + 1] > a[id + i + 2]) {
                int temp = a[id + i + 1];
                a[id + i + 1] = a[id + i + 2];
                a[id + i + 2] = temp;
            }
        }
        __syncthreads();
    }
}

__global__ void radixSort(int* a, int size, int bit) 
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    int sorted[];

    int count[sizeof(int) * 8] = { 0 };

    for (int i = idx; i < size; i += stride) 
    {
        ++count[(a[i] >> bit) & (sizeof(int) * 8 - 1)];
    }
    __syncthreads();

    for (int i = size - 1; i >= 0; i -= stride) 
    {
        sorted[count[(a[i] >> bit) & (sizeof(int) * 8 - 1)] - 1] = a[i];
        --count[(a[i] >> bit) & (sizeof(int) * 8 - 1)];
    }
    __syncthreads();

    for (int i = idx; i < size; i += stride) 
    {
        a[i] = sorted[i];
    }
}

void evenOdd(int* array, int size)
{
    int* a;

    hipMalloc(&a, size * sizeof(int));
    hipMemcpy(a, array, size * sizeof(int), hipMemcpyHostToDevice);

    evenOddSort<<<2, size / 2>>> (a, size);

    hipMemcpy(array, a, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(a);
}

void radix(int* array, int size)
{
    int* a;

    hipMalloc(&a, size * sizeof(int));
    hipMemcpy(a, array, size * sizeof(int), hipMemcpyHostToDevice);

    for (int bit = 0; bit < sizeof(int) * 8; ++bit) {
        radixSort<<<sizeof(int) * 8, size>>> (a, size, bit);
        hipDeviceSynchronize();
    }

    hipMemcpy(array, a, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(a);
}

int main() {
    // Nem biztos, hogy mukodik, mivel otthon megfelelo videokartya hianyaban nem tudtam kiprobalni

    const int size = 5;
    int array[size] = { 12, 5, 3, 4, 1};
    
    //evenOdd(array, size);
    //radix(array, size);

    for (int i = 0; i < size; i++) {
        printf("%d ", array[i]);
    }
    
    return 0;
}
